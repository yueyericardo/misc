#include "hip/hip_runtime.h"
#include <torch/extension.h>
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/ceil_div.h>
#include <ATen/core/TensorBase.h>
#include <ATen/cuda/ScanUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/native/cuda/SortUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/SortingRadixSelect.cuh>

#include <c10/macros/Macros.h>

using at::round_up;
using at::TensorBase;
using torch::Tensor;
using namespace at::native;

namespace {

template <typename T>
struct AddOp {
  __device__ __forceinline__ T operator()(T const& lhs, T const& rhs) {
    return (lhs + rhs);
  }
};

template <typename T, typename IndexType, int Dim, bool Order>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void gatherTopK(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    IndexType inputSliceSize,
    IndexType outputSliceSize, // aka `k`

    IndexType numInputSlices,
    IndexType inputWithinSliceStride,

    at::cuda::detail::TensorInfo<T, IndexType> topK,
    IndexType numTopKSlices,
    IndexType topKWithinSliceStride,

    at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
    IndexType indicesWithinSliceStride) {
  // Indices are limited to integer fp precision, so counts can fit in
  // int32, regardless of IndexType
#if defined(USE_ROCM)
  __shared__ int smem[64];
#else
  __shared__ int smem[32]; // one per each warp, up to warp limit
#endif
  IndexType slice = getLinearBlockId<IndexType>();
  if (slice >= numInputSlices) {
    return;
  }

  // Find the start offset for our slice
  IndexType sliceStartIndex = at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, input);
  IndexType topKSliceStartIndex = at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, topK);
  IndexType indicesSliceStartIndex = at::cuda::detail::IndexToOffset<int64_t, IndexType, Dim>::get(slice, indices);

  T* inputSliceStart = &input.data[sliceStartIndex];
  T* topKSliceStart = &topK.data[topKSliceStartIndex];
  int64_t* indicesSliceStart = &indices.data[indicesSliceStartIndex];

  // Find the k-th highest element in our input
  T topKValue = static_cast<T>(0);
  radixSelect<T, typename TopKTypeConfig<T>::RadixType, IndexType, Order>(
      inputSliceStart, outputSliceSize, inputSliceSize, inputWithinSliceStride, smem, &topKValue);
  const auto topKConverted = at::native::TopKTypeConfig<T>::convert(topKValue);

  // Every value that is strictly less/greater than `pattern`
  // (depending on sort dir) in sorted int format is in the top-K.
  // The top-K value itself might not be unique.
  //
  // Since there are a variable number of elements that we see that
  // are within the top-k, we don't know at what index to write out
  // the resulting values.
  // In order to get this, we perform an exclusive prefix sum of
  // `hasTopK`. This will return the resulting index into which we
  // need to write the result, if a thread has a result.

  // All threads need to participate in the loop and the prefix sum,
  // but not necessarily in the load; hence loop bounds being rounded
  // up to a multiple of the block dim.
  IndexType numIterations = round_up(inputSliceSize, (IndexType)blockDim.x);
  IndexType writeIndexStart = 0;

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v = inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK;
    if (Order) {
      hasTopK = inRange && (convertedV > topKConverted);
    } else {
      hasTopK = inRange && (convertedV < topKConverted);
    }

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    writeIndexStart += carry;
  }

  // We need to fill in the rest with actual == top-K values.
  // The number that we need is outputSliceSize -
  // writeIndexStart. There might be more than that number available,
  // in which case we have to choose the first seen set. We do this
  // via a prefix sum to calculate indices for writing results.
  CUDA_KERNEL_ASSERT(outputSliceSize >= writeIndexStart);
  IndexType topKRemaining = (outputSliceSize - writeIndexStart);

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v = inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK = inRange && (convertedV == topKConverted);

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK && index < topKRemaining) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    if (carry >= topKRemaining) {
      break;
    }

    topKRemaining -= carry;
    writeIndexStart += carry;
  }
};

} // namespace

template <typename T, typename IndexType, int Dim, bool Order>
void dispatchGatherTopK(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    IndexType inputSliceSize,
    IndexType outputSliceSize, // aka `k`

    IndexType numInputSlices,
    IndexType inputWithinSliceStride,

    at::cuda::detail::TensorInfo<T, IndexType> topK,
    IndexType numTopKSlices,
    IndexType topKWithinSliceStride,

    at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
    IndexType indicesWithinSliceStride) {
  dim3 grid;
  TORCH_INTERNAL_ASSERT(getGridFromTiles(numInputSlices, grid), "Too many slices to sort");
  dim3 block(
      std::min(at::ceil_div((int64_t)inputSliceSize, (int64_t)C10_WARP_SIZE) * (int64_t)C10_WARP_SIZE, (int64_t)1024));

  gatherTopK<T, IndexType, Dim, Order><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
      input,
      inputSliceSize,
      outputSliceSize,
      numInputSlices,
      inputWithinSliceStride,
      topK,
      numTopKSlices,
      topKWithinSliceStride,
      indices,
      indicesWithinSliceStride);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void launch_gather_topk_kernel(
    const Tensor& self,
    int64_t k,
    int64_t dim,
    bool largest,
    bool sorted,
    const Tensor& values,
    const Tensor& indices) {
  int numDims = self.dim();
  numDims = numDims == 0 ? 1 : numDims;
  TORCH_CHECK(numDims <= MAX_DIMS, "input tensor has too many dimensions");
  int64_t sliceSize = self.dim() == 0 ? 1 : self.size(dim);

  auto input = self.contiguous();
  // static_cast is required to ensure that the correct type (INDEX_T)
  // is provided to the kernel for the arguments.

#define RUN_K(INDEX_T, DIM, DIR)                                                                                     \
  dispatchGatherTopK<scalar_t, INDEX_T, DIM, DIR>(                                                                   \
      inputInfo,                                                                                                     \
      static_cast<INDEX_T>(sliceSize),                                                                               \
      static_cast<INDEX_T>(k),                                                                                       \
      static_cast<INDEX_T>(inputSlices), /* The actual dimension that the k-selection is running in may have changed \
                                            from collapseDims() */                                                   \
      static_cast<INDEX_T>(inputInfo.strides[collapseInputDim]),                                                     \
      topKInfo,                                                                                                      \
      static_cast<INDEX_T>(topKSlices),                                                                              \
      static_cast<INDEX_T>(topKInfo.strides[collapseTopKDim]),                                                       \
      indicesInfo,                                                                                                   \
      static_cast<INDEX_T>(indicesInfo.strides[collapseIndicesDim]));

#define RUN_DIR(INDEX_T, DIM)   \
  if (largest) {                \
    RUN_K(INDEX_T, DIM, true);  \
  } else {                      \
    RUN_K(INDEX_T, DIM, false); \
  }

#define RUN_DIM(INDEX_T)     \
  if (allDims == 1) {        \
    RUN_DIR(INDEX_T, 1);     \
  } else if (allDims == 2) { \
    RUN_DIR(INDEX_T, 2);     \
  } else if (allDims == 3) { \
    RUN_DIR(INDEX_T, 3);     \
  } else {                   \
    RUN_DIR(INDEX_T, -1);    \
  }

#define RUN_T(INDEX_T)                                                     \
  do {                                                                     \
    using scalar_t = float;                                                \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> inputInfo =            \
        at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(input);         \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> topKInfo =             \
        at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(values);        \
    at::cuda::detail::TensorInfo<int64_t, INDEX_T> indicesInfo =           \
        at::cuda::detail::getTensorInfo<int64_t, INDEX_T>(indices);        \
    /* tensorInfoLegacyIfScalar*/                                          \
    if (!input.dim()) {                                                    \
      inputInfo.dims = 1;                                                  \
      inputInfo.sizes[0] = 1;                                              \
      inputInfo.strides[0] = 1;                                            \
      topKInfo.dims = 1;                                                   \
      topKInfo.sizes[0] = 1;                                               \
      topKInfo.strides[0] = 1;                                             \
      indicesInfo.dims = 1;                                                \
      indicesInfo.sizes[0] = 1;                                            \
      indicesInfo.strides[0] = 1;                                          \
    }                                                                      \
    /* We use these structures solely to find the offset to */             \
    /* each slice we are operating on */                                   \
    inputInfo.sizes[dim] = 1;                                              \
    topKInfo.sizes[dim] = 1;                                               \
    indicesInfo.sizes[dim] = 1;                                            \
    /* stash the stride of dim because it can be accidentally collapsed */ \
    auto strideTopK = topKInfo.strides[dim];                               \
    auto strideIndices = indicesInfo.strides[dim];                         \
    /* Collapse all other dims */                                          \
    int collapseInputDim = inputInfo.collapseDims(dim);                    \
    int collapseTopKDim = topKInfo.collapseDims(dim);                      \
    int collapseIndicesDim = indicesInfo.collapseDims(dim);                \
    /* restore stride in case it was collapsed */                          \
    topKInfo.strides[collapseTopKDim] = strideTopK;                        \
    indicesInfo.strides[collapseIndicesDim] = strideIndices;               \
    int64_t inputSlices = 1;                                               \
    for (int i = 0; i < inputInfo.dims; ++i) {                             \
      inputSlices *= inputInfo.sizes[i];                                   \
    }                                                                      \
    int64_t topKSlices = 1;                                                \
    for (int i = 0; i < topKInfo.dims; ++i) {                              \
      topKSlices *= topKInfo.sizes[i];                                     \
    }                                                                      \
                                                                           \
    /* This is used as a template parameter to calculate indices. */       \
    /* We only specialize it if all collapsed dim sizes are the */         \
    /* same; otherwise, we use -1 which is the specialization */           \
    /* parameter for arbitrary dimensions */                               \
    int allDims = inputInfo.dims;                                          \
    if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {         \
      allDims = -1;                                                        \
    }                                                                      \
                                                                           \
    RUN_DIM(INDEX_T);                                                      \
  } while (0)

  // the below is safe with 0-dimensional tensors because it is based on
  // TensorInfo which implicitly expands to 1-dimensional.
  if (input.numel() > 0) {
    // Based on required index size, run the algorithm with the
    // appropriate index type
    if (at::cuda::detail::canUse32BitIndexMath(input) && at::cuda::detail::canUse32BitIndexMath(values) &&
        at::cuda::detail::canUse32BitIndexMath(indices)) {
      RUN_T(uint32_t);
    } else {
      RUN_T(uint64_t);
    }
  }
#undef RUN_T
#undef RUN_DIM
#undef RUN_DIR
#undef RUN_K
}

TORCH_LIBRARY(mbtopk, m) {
  m.def("multiBlockTopK", launch_gather_topk_kernel);
}

TORCH_LIBRARY_IMPL(mbtopk, CUDA, m) {
  m.impl("multiBlockTopK", launch_gather_topk_kernel);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {}
