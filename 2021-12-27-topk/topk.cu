#include "hip/hip_runtime.h"
#include <torch/extension.h>
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/ceil_div.h>
#include <ATen/core/TensorBase.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/cuda/AsmUtils.cuh>
#include <ATen/cuda/DeviceUtils.cuh>
#include <ATen/cuda/ScanUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/native/cuda/SortUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/SortingRadixSelect.cuh>
#include <cub/block/block_scan.cuh>
#include <iostream>

#include <c10/macros/Macros.h>

using at::round_up;
using at::TensorBase;
using torch::Tensor;
using namespace at::native;

namespace mbtopk {

template <typename T>
struct AddOp {
  __device__ __forceinline__ T operator()(T const& lhs, T const& rhs) {
    return (lhs + rhs);
  }
};

// A stateful callback functor that maintains a running prefix to be applied
// during consecutive scan operations.
struct BlockPrefixCallbackOp {
  // Running prefix
  int running_total;
  // Constructor
  __device__ BlockPrefixCallbackOp(int running_total) : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide scan.
  __device__ int operator()(int block_aggregate) {
    int old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T>
__global__ void print_kernel(T* x) {
  printf("%f\n", *x);
}

template <typename T>
__global__ void fill(T* x, T value, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < size; i += gridDim.x * blockDim.x) {
    x[i] = value;
  }
}

template <typename T, typename IndexType, int Dim, bool Order>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void gatherTopK(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    IndexType inputSliceSize,
    IndexType outputSliceSize, // aka `k`

    IndexType numInputSlices,
    IndexType inputWithinSliceStride,

    at::cuda::detail::TensorInfo<T, IndexType> topK,
    IndexType numTopKSlices,
    IndexType topKWithinSliceStride,

    at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
    IndexType indicesWithinSliceStride,
    T* kthValues) {
  // Indices are limited to integer fp precision, so counts can fit in
  // int32, regardless of IndexType
#if defined(USE_ROCM)
  __shared__ int smem[64];
#else
  __shared__ int smem[32]; // one per each warp, up to warp limit
#endif
  IndexType slice = getLinearBlockId<IndexType>();
  if (slice >= numInputSlices) {
    return;
  }

  // Find the start offset for our slice
  IndexType sliceStartIndex = at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, input);
  IndexType topKSliceStartIndex = at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, topK);
  IndexType indicesSliceStartIndex = at::cuda::detail::IndexToOffset<int64_t, IndexType, Dim>::get(slice, indices);

  T* inputSliceStart = &input.data[sliceStartIndex];
  T* topKSliceStart = &topK.data[topKSliceStartIndex];
  int64_t* indicesSliceStart = &indices.data[indicesSliceStartIndex];

  // Find the k-th highest element in our input
  T topKValue = kthValues[slice];
  const auto topKConverted = at::native::TopKTypeConfig<T>::convert(topKValue);

  // Every value that is strictly less/greater than `pattern`
  // (depending on sort dir) in sorted int format is in the top-K.
  // The top-K value itself might not be unique.
  //
  // Since there are a variable number of elements that we see that
  // are within the top-k, we don't know at what index to write out
  // the resulting values.
  // In order to get this, we perform an exclusive prefix sum of
  // `hasTopK`. This will return the resulting index into which we
  // need to write the result, if a thread has a result.

  // All threads need to participate in the loop and the prefix sum,
  // but not necessarily in the load; hence loop bounds being rounded
  // up to a multiple of the block dim.
  IndexType numIterations = round_up(inputSliceSize, (IndexType)blockDim.x);
  IndexType writeIndexStart = 0;

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v = inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK;
    if (Order) {
      hasTopK = inRange && (convertedV > topKConverted);
    } else {
      hasTopK = inRange && (convertedV < topKConverted);
    }

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    writeIndexStart += carry;
  }

  // We need to fill in the rest with actual == top-K values.
  // The number that we need is outputSliceSize -
  // writeIndexStart. There might be more than that number available,
  // in which case we have to choose the first seen set. We do this
  // via a prefix sum to calculate indices for writing results.
  CUDA_KERNEL_ASSERT(outputSliceSize >= writeIndexStart);
  IndexType topKRemaining = (outputSliceSize - writeIndexStart);

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v = inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK = inRange && (convertedV == topKConverted);

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK && index < topKRemaining) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    if (carry >= topKRemaining) {
      break;
    }

    topKRemaining -= carry;
    writeIndexStart += carry;
  }
};

constexpr int BLOCK_THREADS = 128;
// in principle, we could write at most 255 into digit counter (in shared mem) with unsigned char type
// TODO tune this, maybe smaller
constexpr int MAX_ITEMS_PER_THREAD = 64;
constexpr int ITEMS_PER_BLOCK = BLOCK_THREADS * MAX_ITEMS_PER_THREAD;

// Over what radix we are selecting values
constexpr int RADIX_BITS = 6; // digits are base-(2 ^ RADIX_BITS)
constexpr int RADIX_DIGITS = 1 << RADIX_BITS; // 2 ^ RADIX_BITS
constexpr int RADIX_MASK = (RADIX_DIGITS - 1);

template <typename T, typename IndexType, typename Bitwise, int Dim, bool Order, int RADIX_BITS>
C10_LAUNCH_BOUNDS_1(BLOCK_THREADS)
__global__ void radixFindKthValues(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    IndexType inputSliceSize,
    IndexType* ksToFind, // aka `k`

    IndexType numInputSlices,
    IndexType withinSliceStride,

    int current_bit,
    IndexType blocks_per_slice,
    Bitwise desiredMask,

    // outputs
    int* semaphores,
    Bitwise* desires,
    IndexType* counts,
    T* kthValues // only writes when current_bit reaches 0
) {
  constexpr int RADIX_DIGITS = 1 << RADIX_BITS; // 2 ^ RADIX_BITS
  constexpr int PACKING_RATIO = sizeof(int) / sizeof(unsigned char);
  constexpr int COUNTER_LANES = RADIX_DIGITS / PACKING_RATIO;

  int tidx = threadIdx.x;
  IndexType block_idx = getLinearBlockId<IndexType>();
  IndexType slice_idx = block_idx / blocks_per_slice;
  IndexType blk_idx_in_slice = block_idx % blocks_per_slice;
  if (slice_idx >= numInputSlices) {
    return;
  }

  // printf("Reach line number %d, %d %d %d | %d %d \n", __LINE__, block_idx, slice_idx, tidx, current_bit,
  // semaphores[slice_idx]);
  Bitwise desired = desires[slice_idx];
  IndexType kToFind = ksToFind[slice_idx];
  IndexType sliceStartIndex = at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice_idx, input);
  T* data = &input.data[sliceStartIndex];

  typedef hipcub::BlockScan<IndexType, BLOCK_THREADS> BlockScan;
  union __align__(16) TempStorage {
    unsigned char thread_counters[COUNTER_LANES][BLOCK_THREADS]
                                 [PACKING_RATIO]; // threads in a warp is guaranteed to access different banks
    uint32_t packed_thread_counters[COUNTER_LANES][BLOCK_THREADS];
    struct {
      IndexType digit_count_cumsum[RADIX_DIGITS];
      typename BlockScan::TempStorage temp_storage;
    } scan_storage;
  };
  __shared__ TempStorage temp_storage;

  // reset temp_storage
  for (int i = 0; i < COUNTER_LANES; ++i) {
    temp_storage.packed_thread_counters[i][tidx] = 0;
  }
  __syncthreads();

  int items_per_thread = (blk_idx_in_slice + 1 < blocks_per_slice)
      ? MAX_ITEMS_PER_THREAD
      : at::ceil_div((int64_t)(inputSliceSize - blk_idx_in_slice * ITEMS_PER_BLOCK), (int64_t)BLOCK_THREADS);

  // collect counts and store in shared memorey for each thread
  for (int i = 0; i < items_per_thread; ++i) {
    // Find the start offset for our slice
    IndexType idx = (tidx + i * BLOCK_THREADS + blk_idx_in_slice * ITEMS_PER_BLOCK) * withinSliceStride;
    if (idx < inputSliceSize) {
      T val_ori = doLdg(&data[idx]);
      Bitwise val = TopKTypeConfig<T>::convert(doLdg(&data[idx]));
      bool hasVal = ((val & desiredMask) == (desired & desiredMask));
      // printf("Reach line number %d, %d %d %d | %ud %x %ud %x\n", __LINE__, block_idx, slice_idx, tidx, val, val,
      // desired, desired);
      if (3 < val_ori && val_ori < 4) {
        // printf("Reach line number %d, %d %d %d | %f| %ud %x %ud %x | %d\n", __LINE__, block_idx, slice_idx, tidx,
        // val_ori, val, val, desired, desired, current_bit);
      }
      Bitwise digit = at::cuda::Bitfield<Bitwise>::getBitfield(val, current_bit, RADIX_BITS);
      if (hasVal) {
        temp_storage
            .thread_counters[digit / PACKING_RATIO][tidx]
                            [digit % PACKING_RATIO]++; // threads in a warp is guaranteed to access different banks
        int a = temp_storage.thread_counters[digit / PACKING_RATIO][tidx][digit % PACKING_RATIO];
        // printf("Reach line number %d, %d %d %d | %d %x %d\n", __LINE__, block_idx, slice_idx, tidx, digit, digit, a);
      }
    }
  }

  __syncthreads();

  // extract counts and write count out
  for (int i = 0; i < (RADIX_DIGITS + BLOCK_THREADS - 1) / BLOCK_THREADS; ++i) {
    // every thread collects one overall digit count stored in shared mem for each thread
    int digit_count = 0;
    int digit = i * BLOCK_THREADS + tidx;
    if (digit < RADIX_DIGITS) {
      for (int j = 0, idx = tidx; j < BLOCK_THREADS;
           ++j, idx = (idx + 1) % BLOCK_THREADS) { // every thread access different bank
        digit_count += temp_storage.thread_counters[digit / PACKING_RATIO][idx][digit % PACKING_RATIO];
        int a = temp_storage.thread_counters[digit / PACKING_RATIO][idx][digit % PACKING_RATIO];
        if (a > 0) {
          // printf("Reach line number %d, %d %d %d | %d %x %d %d\n", __LINE__, block_idx, slice_idx, tidx, digit,
          // digit, a, digit_count);
        }
      }
      counts[block_idx * RADIX_DIGITS + digit] = digit_count;
    }
    if (digit_count > 0) {
      // printf("Reach line number %d, %d %d %d | %x %d\n", __LINE__, block_idx, slice_idx, tidx, digit, digit_count);
    }
  }

  __threadfence(); // make sure writes are globally visible
  __syncthreads();

  __shared__ bool s_is_last_block_done;
  __shared__ bool s_desired_found;

  if (tidx == 0) {
    int blocks_finished_old = atomicAdd(&semaphores[slice_idx], 1);
    s_is_last_block_done = (blocks_finished_old == blocks_per_slice - 1);
    s_desired_found = false;
  }

  __syncthreads();

  // last block for each slice accumulate counts from blocks and update desired
  if (s_is_last_block_done) {
    // sum block counts
    BlockPrefixCallbackOp prefix_op(0);

    auto post_process = [&](int digit) {
      IndexType digit_count = 0;
      IndexType& digit_count_cumsum = digit_count;
      for (int blk = 0; blk < blocks_per_slice; ++blk) {
        digit_count += counts[(slice_idx * blocks_per_slice + blk) * RADIX_DIGITS + digit];
      }

      // Collectively compute the block-wide exclusive prefix sum
      BlockScan(temp_storage.scan_storage.temp_storage).InclusiveSum(digit_count, digit_count_cumsum, prefix_op);
      __syncthreads();
      temp_storage.scan_storage.digit_count_cumsum[digit] = digit_count_cumsum;
      __syncthreads();

      // update desired
      IndexType digit_count_cumsum_left;
      if (Order) {
        digit_count_cumsum_left =
            (digit == RADIX_DIGITS - 1) ? 0 : temp_storage.scan_storage.digit_count_cumsum[digit + 1];
      } else {
        digit_count_cumsum_left = (digit == 0) ? 0 : temp_storage.scan_storage.digit_count_cumsum[digit - 1];
      }
      // printf("Reach line number %d, %d %d %d | %d %x | %d %d %d \n", __LINE__, block_idx, slice_idx, tidx, digit,
      // digit, kToFind, digit_count_cumsum_left, digit_count_cumsum);
      if (digit_count_cumsum_left < kToFind && kToFind <= digit_count_cumsum) {
        desired = at::cuda::Bitfield<Bitwise>::setBitfield(desired, digit, current_bit, RADIX_BITS);
        // printf("Reach line number %d, %d %d %d | %d %x %d %x | %d\n", __LINE__, block_idx, slice_idx, tidx,
        // (int)digit, (int)digit, desired, desired, kToFind);
        if (current_bit > 0) {
          desires[slice_idx] = desired;
          ksToFind[slice_idx] = kToFind - digit_count_cumsum_left;
          int a = kToFind - digit_count_cumsum_left;
          // printf("Reach line number %d, %d %d %d | %ud %x %ud %x | %d %d | %d\n", __LINE__, block_idx, slice_idx,
          // tidx, (int)digit, (int)digit, desired, desired, kToFind, a, current_bit);
        } else {
          kthValues[slice_idx] = TopKTypeConfig<T>::deconvert(desired);
          // printf("1 %d %d %d %f\n", block_idx, slice_idx, tidx, TopKTypeConfig<T>::deconvert(desired));
        }
        s_desired_found = true;
      }
      __syncthreads();
    };

    if (Order) {
      for (int digit = RADIX_DIGITS - tidx - 1; digit >= 0 && !s_desired_found; digit -= BLOCK_THREADS) {
        post_process(digit);
      }
    } else {
      for (int digit = tidx; digit < RADIX_DIGITS && !s_desired_found; digit += BLOCK_THREADS) {
        post_process(digit);
      }
    }

    if (tidx == 0) {
      semaphores[slice_idx] = 0;
    }
  }
};

// TODO renmae Order to IsAscending
template <typename T, typename IndexType, int Dim, bool Order, bool multiblock = true>
void dispatchGatherTopK(
    at::cuda::detail::TensorInfo<T, IndexType> input,
    // TODO sizeof TensorInfo (216 bytes) is very big, which is not necessary
    IndexType inputSliceSize,
    IndexType outputSliceSize, // aka `k`

    IndexType numInputSlices,
    IndexType inputWithinSliceStride,

    at::cuda::detail::TensorInfo<T, IndexType> topK,
    IndexType numTopKSlices, // TODO never used
    IndexType topKWithinSliceStride,

    at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
    IndexType indicesWithinSliceStride) {
  using Bitwise = typename TopKTypeConfig<T>::RadixType;
  // std::cout << "sizeof tensorinfo: " << sizeof(at::cuda::detail::TensorInfo<T, IndexType>) << std::endl;
  // std::cout << "sizeof int: " << sizeof(int) << std::endl;
  if (multiblock) {
    int64_t blocks_per_slice = at::ceil_div((int64_t)inputSliceSize, (int64_t)ITEMS_PER_BLOCK);
    int64_t num_blocks = numInputSlices * blocks_per_slice;

    // temporary storage
    auto& allocator = *c10::cuda::CUDACachingAllocator::get();

    auto kthValues_buffer = allocator.allocate(numInputSlices * sizeof(T));
    T* kthValues = reinterpret_cast<T*>(kthValues_buffer.get());
    AT_CUDA_CHECK(hipMemsetAsync(kthValues, 0, numInputSlices * sizeof(T), c10::cuda::getCurrentCUDAStream()));

    auto semaphores_buffer = allocator.allocate(numInputSlices * sizeof(int));
    int* semaphores = reinterpret_cast<int*>(semaphores_buffer.get());
    AT_CUDA_CHECK(hipMemsetAsync(semaphores, 0, numInputSlices * sizeof(int), c10::cuda::getCurrentCUDAStream()));

    auto ksToFind_buffer = allocator.allocate(numInputSlices * sizeof(IndexType));
    IndexType* ksToFind = reinterpret_cast<IndexType*>(ksToFind_buffer.get());
    fill<IndexType>
        <<<std::min((numInputSlices + 511) / 512, (IndexType)65535), 512, 0, c10::cuda::getCurrentCUDAStream()>>>(
            ksToFind, outputSliceSize, numInputSlices);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    auto desired_buffer = allocator.allocate(numInputSlices * sizeof(Bitwise));
    Bitwise* desired = reinterpret_cast<Bitwise*>(desired_buffer.get());

    auto counts_buffer = allocator.allocate(num_blocks * RADIX_DIGITS * sizeof(IndexType));
    IndexType* counts = reinterpret_cast<IndexType*>(counts_buffer.get());

    Bitwise desiredMask = 0;
    dim3 grid;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(num_blocks, grid), "Too many slices to sort");
    dim3 block(BLOCK_THREADS);

#define RUN_K(BIT)                                                                                                   \
  radixFindKthValues<T, IndexType, Bitwise, Dim, Order, BIT><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>( \
      input,                                                                                                         \
      inputSliceSize,                                                                                                \
      ksToFind,                                                                                                      \
      numInputSlices,                                                                                                \
      inputWithinSliceStride,                                                                                        \
      current_bit,                                                                                                   \
      blocks_per_slice,                                                                                              \
      desiredMask,                                                                                                   \
      semaphores,                                                                                                    \
      desired,                                                                                                       \
      counts,                                                                                                        \
      kthValues);                                                                                                    \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define RUN_BIT()                                                                \
  if (radix_bits == 6) {                                                         \
    RUN_K(6);                                                                    \
  } else if (radix_bits == 4) {                                                  \
    RUN_K(4);                                                                    \
  } else if (radix_bits == 2) {                                                  \
    RUN_K(2);                                                                    \
  } else {                                                                       \
    TORCH_INTERNAL_ASSERT(false, "RADIX_BIT ", radix_bits, " is not supported"); \
  }

    int current_bit = sizeof(T) * 8 - RADIX_BITS;
    int radix_bits = RADIX_BITS;
    for (; current_bit > 0; current_bit -= RADIX_BITS) {
      // std::cout << "current_bit: " << current_bit << std::endl;
      // std::cout << "desiredMask: " << std::bitset<32>(desiredMask).to_string() << std::endl;
      RUN_BIT();
      desiredMask = at::cuda::Bitfield<Bitwise>::setBitfield(desiredMask, RADIX_MASK, current_bit, RADIX_BITS);
    }
    radix_bits = current_bit + RADIX_BITS;
    current_bit = 0;
    RUN_BIT();

#undef RUN_BIT
#undef RUN_K

    // for (int i = 0; i < numInputSlices; ++i) {
    //   std::cout << "slice " << i << ": ";
    //   hipStreamSynchronize(c10::cuda::getCurrentCUDAStream());
    //   print_kernel<T><<<1, 1>>>(kthValues + i);
    //   hipStreamSynchronize(c10::cuda::getCurrentCUDAStream());
    // }

    // Find values that are strictly less/greater than the top-K value

    // Find values that are == the top-K value
    if (true) {
      dim3 grid;
      TORCH_INTERNAL_ASSERT(getGridFromTiles(numInputSlices, grid), "Too many slices to sort");
      dim3 block(std::min(
          at::ceil_div((int64_t)inputSliceSize, (int64_t)C10_WARP_SIZE) * (int64_t)C10_WARP_SIZE, (int64_t)1024));
      gatherTopK<T, IndexType, Dim, Order><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
          input,
          inputSliceSize,
          outputSliceSize,
          numInputSlices,
          inputWithinSliceStride,
          topK,
          numTopKSlices,
          topKWithinSliceStride,
          indices,
          indicesWithinSliceStride,
          kthValues);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  } else {
    ;
    // dim3 grid;
    // TORCH_INTERNAL_ASSERT(getGridFromTiles(numInputSlices, grid), "Too many slices to sort");
    // dim3 block(std::min(
    //     at::ceil_div((int64_t)inputSliceSize, (int64_t)C10_WARP_SIZE) * (int64_t)C10_WARP_SIZE, (int64_t)1024));
    // gatherTopK<T, IndexType, Dim, Order><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
    //     input,
    //     inputSliceSize,
    //     outputSliceSize,
    //     numInputSlices,
    //     inputWithinSliceStride,
    //     topK,
    //     numTopKSlices,
    //     topKWithinSliceStride,
    //     indices,
    //     indicesWithinSliceStride);
    // C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

} // namespace mbtopk

void launch_gather_topk_kernel(
    const Tensor& self,
    int64_t k,
    int64_t dim,
    bool largest,
    bool sorted,
    const Tensor& values,
    const Tensor& indices) {
  int numDims = self.dim();
  numDims = numDims == 0 ? 1 : numDims;
  TORCH_CHECK(numDims <= MAX_DIMS, "input tensor has too many dimensions");
  int64_t sliceSize = self.dim() == 0 ? 1 : self.size(dim);

  auto input = self.contiguous();
  // static_cast is required to ensure that the correct type (INDEX_T)
  // is provided to the kernel for the arguments.

#define RUN_K(INDEX_T, DIM, DIR)                                                                                     \
  mbtopk::dispatchGatherTopK<scalar_t, INDEX_T, DIM, DIR>(                                                           \
      inputInfo,                                                                                                     \
      static_cast<INDEX_T>(sliceSize),                                                                               \
      static_cast<INDEX_T>(k),                                                                                       \
      static_cast<INDEX_T>(inputSlices), /* The actual dimension that the k-selection is running in may have changed \
                                            from collapseDims() */                                                   \
      static_cast<INDEX_T>(inputInfo.strides[collapseInputDim]),                                                     \
      topKInfo,                                                                                                      \
      static_cast<INDEX_T>(topKSlices),                                                                              \
      static_cast<INDEX_T>(topKInfo.strides[collapseTopKDim]),                                                       \
      indicesInfo,                                                                                                   \
      static_cast<INDEX_T>(indicesInfo.strides[collapseIndicesDim]));

#define RUN_DIR(INDEX_T, DIM)   \
  if (largest) {                \
    RUN_K(INDEX_T, DIM, true);  \
  } else {                      \
    RUN_K(INDEX_T, DIM, false); \
  }

#define RUN_DIM(INDEX_T)     \
  if (allDims == 1) {        \
    RUN_DIR(INDEX_T, 1);     \
  } else if (allDims == 2) { \
    RUN_DIR(INDEX_T, 2);     \
  } else if (allDims == 3) { \
    RUN_DIR(INDEX_T, 3);     \
  } else {                   \
    RUN_DIR(INDEX_T, -1);    \
  }

#define RUN_T(INDEX_T)                                                     \
  do {                                                                     \
    using scalar_t = float;                                                \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> inputInfo =            \
        at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(input);         \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> topKInfo =             \
        at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(values);        \
    at::cuda::detail::TensorInfo<int64_t, INDEX_T> indicesInfo =           \
        at::cuda::detail::getTensorInfo<int64_t, INDEX_T>(indices);        \
    /* tensorInfoLegacyIfScalar*/                                          \
    if (!input.dim()) {                                                    \
      inputInfo.dims = 1;                                                  \
      inputInfo.sizes[0] = 1;                                              \
      inputInfo.strides[0] = 1;                                            \
      topKInfo.dims = 1;                                                   \
      topKInfo.sizes[0] = 1;                                               \
      topKInfo.strides[0] = 1;                                             \
      indicesInfo.dims = 1;                                                \
      indicesInfo.sizes[0] = 1;                                            \
      indicesInfo.strides[0] = 1;                                          \
    }                                                                      \
    /* We use these structures solely to find the offset to */             \
    /* each slice we are operating on */                                   \
    inputInfo.sizes[dim] = 1;                                              \
    topKInfo.sizes[dim] = 1;                                               \
    indicesInfo.sizes[dim] = 1;                                            \
    /* stash the stride of dim because it can be accidentally collapsed */ \
    auto strideTopK = topKInfo.strides[dim];                               \
    auto strideIndices = indicesInfo.strides[dim];                         \
    /* Collapse all other dims */                                          \
    int collapseInputDim = inputInfo.collapseDims(dim);                    \
    int collapseTopKDim = topKInfo.collapseDims(dim);                      \
    int collapseIndicesDim = indicesInfo.collapseDims(dim);                \
    /* restore stride in case it was collapsed */                          \
    topKInfo.strides[collapseTopKDim] = strideTopK;                        \
    indicesInfo.strides[collapseIndicesDim] = strideIndices;               \
    int64_t inputSlices = 1;                                               \
    for (int i = 0; i < inputInfo.dims; ++i) {                             \
      inputSlices *= inputInfo.sizes[i];                                   \
    }                                                                      \
    int64_t topKSlices = 1;                                                \
    for (int i = 0; i < topKInfo.dims; ++i) {                              \
      topKSlices *= topKInfo.sizes[i];                                     \
    }                                                                      \
                                                                           \
    /* This is used as a template parameter to calculate indices. */       \
    /* We only specialize it if all collapsed dim sizes are the */         \
    /* same; otherwise, we use -1 which is the specialization */           \
    /* parameter for arbitrary dimensions */                               \
    int allDims = inputInfo.dims;                                          \
    if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {         \
      allDims = -1;                                                        \
    }                                                                      \
                                                                           \
    RUN_DIM(INDEX_T);                                                      \
  } while (0)

  // the below is safe with 0-dimensional tensors because it is based on
  // TensorInfo which implicitly expands to 1-dimensional.
  if (input.numel() > 0) {
    // Based on required index size, run the algorithm with the
    // appropriate index type
    if (at::cuda::detail::canUse32BitIndexMath(input) && at::cuda::detail::canUse32BitIndexMath(values) &&
        at::cuda::detail::canUse32BitIndexMath(indices)) {
      RUN_T(uint32_t);
    } else {
      RUN_T(uint64_t);
    }
  }
#undef RUN_T
#undef RUN_DIM
#undef RUN_DIR
#undef RUN_K
}

TORCH_LIBRARY(mbtopk, m) {
  m.def("multiBlockTopK", launch_gather_topk_kernel);
}

TORCH_LIBRARY_IMPL(mbtopk, CUDA, m) {
  m.impl("multiBlockTopK", launch_gather_topk_kernel);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {}
